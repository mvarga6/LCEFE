//=============================================================//
//                                                             //
//            ||Gpu Accelerated Fineite Element ||             //
//                                                             //
//              --------Version 6.0----------                  //
//                                                             //
//                                                             //
//                                                             //
//    Authors: Andrew Konya      (Kent State University)       //
//             Robin Selinger    (Kent State University)       // 
//             Badel MBanga      (kent State University)       //
//                                                             //
//   Finite elemnt simulation executed on GPU using CUDA       //
//   Hybrid MD finite element algorithm used to allow          //
//   all computations be implemented locally requireing        //
//   parallelization of all prccess in calculation             //
//                                                             //
//=============================================================//


#include "mainhead.h"


int main(int argc, char *argv[])
{
	//Get commandline arguments
	parseCommandLine(argc, argv);

	//Get Device properties
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop,0));
	printf( "Code executing on %s\n\n", prop.name );
	//displayGPUinfo(prop);

	int Ntets,Nnodes;
	//get dimensions of the mesh
	//get_mesh_dim(Ntets, Nnodes);
	get_gmsh_dim(std::string(MESHFILE), Ntets, Nnodes);

	//create objects of TetArray and NodeArray class with correct size
	TetArray Tet = TetArray(Ntets);
	NodeArray Node = NodeArray(Nnodes);

	//read the mesh into Node and Tet objects
	//get_mesh(Node,Tet,Ntets,Nnodes);
	get_gmsh(std::string(MESHFILE), Node, Tet, Ntets, Nnodes);
	
	const float flatten_Z[3] = {1.0f, 1.0f, 0.75f};
	Node.deform(flatten_Z);
	//Node.eulerRotation(0, PI/2.0, 0);

	//get positions of tetrahedra
	get_tet_pos(Node, Tet, Ntets);

	//set director n for each tetrahedra
	set_n(Tet, Ntets);

	// comment out GPU calculations while Debugging director sim

	//reorder tetrahedra 
	gorder_tet(Node, Tet, Ntets);

	//re-order nodes and reassing tetrahedra component lists
	finish_order(Node, Tet, Ntets, Nnodes);

	//find initial A's and invert them  store all in Tet object
	init_As(Node, Tet, Ntets);

	//print spacefilling curve to represent adjacensy between tetrahedra
	printorder(Tet, Ntets);

	//pritn director
	Tet.printDirector();

	//now ready to prepare for dyanmics
	//delcare data stuctures for data on device
	//and host
	DevDataBlock dev_dat;
	HostDataBlock host_dat;

	//Pack data to send to device
	std::vector<int> surfTets;
	packdata(Node,Tet,&host_dat,Ntets,Nnodes, &surfTets);

	//send data to device
	data_to_device(&dev_dat,&host_dat,Ntets,Nnodes);


	//Print Simulation Parameters and Such
	printf("\n\n Prepared for dynamics with:\n  \
				steps/frame	  =	  %d\n    \
				Volume        =   %f cm^3\n  \
				Mass          =   %f kg\n\n",iterPerFrame,host_dat.host_totalVolume,host_dat.host_totalVolume*materialDensity);




	//=================================================================
	//initillize GPU syncronization arrays
	//will store syncronization information
	//=================================================================
	int Threads_Per_Block = TPB;
	int Blocks = (Ntets+Threads_Per_Block)/Threads_Per_Block;
	int *Syncin,*Syncout,*g_mutex;
	//allocate memory on device for Syncin and Syncoutd

	
	HANDLE_ERROR( hipMalloc( (void**)&Syncin
								,Blocks*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&Syncout
								,Blocks*sizeof(int) ) );

	int* SyncZeros;
	SyncZeros = (int*)malloc(Blocks*sizeof(int));
	for (int i=0;i<Blocks;i++){
		SyncZeros[i]=0;
	}
	
	HANDLE_ERROR( hipMemcpy(Syncin
							,SyncZeros
							,Blocks*sizeof(int)
							,hipMemcpyHostToDevice ) );
	//allocate global mutex and set =0 
	 HANDLE_ERROR( hipMalloc( (void**)&g_mutex,
                              sizeof(int) ) );
     HANDLE_ERROR( hipMemset( g_mutex, 0, sizeof(int) ) );
	 
	//=================================================================
	//run dynamics
	//=================================================================

	run_dynamics(&dev_dat,&host_dat,Ntets,Nnodes,Syncin,Syncout,g_mutex, &surfTets);

	//check for CUDA erros
	any_errors();

	//exit program

	HANDLE_ERROR( hipFree( Syncin ) );
	HANDLE_ERROR(hipFree( Syncout ) );
	HANDLE_ERROR(hipFree( g_mutex ) );
	exit_program(&dev_dat);

	//*/

    return 0;
}
